#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


// Data structures
//
//
// This struct represents a collection of points.
// Arrays x and y represent the position of each point.
// Arrays new and old help us store something related to each point.
// Arrays new_x and new_y are used to calculate new values for the arrays x and y (not always needed).
// struct
// {
//     float *x;
//     float *y;
//     int *new;
//     int *old;
//     float *new_x;
//     float *new_y;
// } typedef points;

// Pointer for all the points with struct points, for the struct points:
// Arrays x and y will be used to store the points.
// Arrays new and old will store the cluster of the point on the present and previous iteration of the algorithm, respectively.
// Arrays new_x and new_y are not needed.
// points *_points_;
float *points_x;
float *points_y;
int *points_cluster;

// Pointer for all the cluster with struct points, for the struct points:
// Arrays x and y will be used to store the clusters center.
// Arrays new and old will store the the number of points in each cluster on the present and previous iteration of the algorithm, respectively.
// Arrays new_x and new_y are used to calculate the center of each cluster  in the next iteration.
// points *clusters_center;
float *clusters_center_x;
float *clusters_center_y;
int *clusters_size;

/**
 * @brief Allocates memory for the data structures.
 *        Generates random points.
 *        Initializes the clusters centers as the first NCLUSTERS points.
 */
void init_ds(int n_points, int n_clusters)
{
    // Allocate memory for all the data structures.
    points_x = (float *)malloc(n_points * sizeof(float));
    points_y = (float *)malloc(n_points * sizeof(float));
    points_cluster = (int *)malloc(n_points * sizeof(int));

    clusters_center_x = (float *)malloc(n_clusters * sizeof(float));
    clusters_center_y = (float *)malloc(n_clusters * sizeof(float));
    clusters_size = (int *)malloc(n_clusters * sizeof(int));

    // Random seed of 10 for reproducibility.
    srand(10);

    // Index variable.
    int i;

    // Generate random points and store them in the points array.
    for (i = 0; i < n_points; i++)
    {
        points_x[i] = (float)rand() / RAND_MAX;
        points_y[i] = (float)rand() / RAND_MAX;
        points_cluster[i] = -1;
    }

    // Initialize the clusters centers as the first NCLUSTERS points.
    for (i = 0; i < n_clusters; i++)
    {
        clusters_center_x[i] = points_x[i];
        clusters_center_y[i] = points_y[i];
    }
}

/**
 * @brief Free previously allocated memmory.
 */
void free_data()
{
    free(points_x);
    free(points_y);
    free(points_cluster);
    free(clusters_center_x);
    free(clusters_center_y);
    free(clusters_size);
}


__global__ void equalToZero(int n_clusters, float *nclusters_center_x, float *nclusters_center_y, int *nclusters_size) {
    // Index variable.
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_clusters)
    {
        nclusters_center_x[i] = 0;
        nclusters_center_y[i] = 0;
        nclusters_size[i] = 0;
    }
}

__global__ void sumClusterPoints(int n_points, int n_clusters, float *npoints_x, float *npoints_y, int *npoints_cluster, float *out_clusters_x, float *out_clusters_y, int *out_clusters_size) {
    extern __shared__ float sdata[];
    float *s_clusters_x = (float *) sdata;
    float *s_clusters_y = n_clusters * blockDim.x * sizeof(float) + sdata;
    int *s_clusters_size = n_clusters * blockDim.x * sizeof(float) + (int *) s_clusters_y;


    // each thread loads data from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    for(int j = 0; j < n_clusters; j++){
        s_clusters_x[tid + j] = 0;
        s_clusters_y[tid + j] = 0;
        s_clusters_size[tid + j] = 0;
    }

    if(i < n_points){
        int cluster = npoints_cluster[i];
        s_clusters_x[tid + cluster] = npoints_x[i];
        s_clusters_y[tid + cluster] = npoints_y[i];
        s_clusters_size[tid + cluster] = 1;

        __syncthreads();

        // do reduction in shared mem
        for(unsigned int s=1; s < blockDim.x; s *= 2) {
            if (tid % (2*s) == 0) {
                for(int j = 0; j < n_clusters; j++){
                    s_clusters_x[tid + j] = s_clusters_x[tid + j + s];
                    s_clusters_y[tid + j] = s_clusters_y[tid + j + s];
                    s_clusters_size[tid + j] = s_clusters_size[tid + j + s];
                }
            }
            __syncthreads();
        }

        // write result for this block to global mem
        if (tid == 0){
            for(int j = 0; j < n_clusters; j++){
                out_clusters_x[blockIdx.x + j] = s_clusters_x[j];
                out_clusters_y[blockIdx.x + j] = s_clusters_y[j];
                out_clusters_size[blockIdx.x + j] = s_clusters_size[j];
            }

        }   
    }
}

__global__ void reduceArrays(int blockNum, int n_clusters, float *out_clusters_x, float *out_clusters_y, int *out_clusters_size) {
    extern __shared__ float data[];
    float * s_clusters_x = (float *) data;
    float * s_clusters_y = n_clusters * blockDim.x * sizeof(float) + data;
    int * s_clusters_size = n_clusters * blockDim.x * sizeof(float) + (int *) s_clusters_y;

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < blockNum) {
        for(int j = 0; j < n_clusters; j++){
            s_clusters_x[tid + j] = out_clusters_x[i + j];
            s_clusters_y[tid + j] = out_clusters_y[i + j];
            s_clusters_size[tid + j] = out_clusters_size[i + j];
        }
        __syncthreads();

        // do reduction in shared mem
        for(unsigned int s=1; s < blockDim.x; s *= 2) {
            if (tid % (2*s) == 0) {
                for(int j = 0; j < n_clusters; j++){
                    s_clusters_x[tid + j] = s_clusters_x[i + s + j];
                    s_clusters_y[tid + j] = s_clusters_y[i + s + j];
                    s_clusters_size[tid + j] = s_clusters_size[i + s + j];
                }
            }
            __syncthreads();
        }

        // write result for this block to global mem
        if (tid == 0){
            for(int j = 0; j < n_clusters; j++){
                out_clusters_x[blockIdx.x + j] = out_clusters_x[j];
                out_clusters_y[blockIdx.x + j] = out_clusters_y[j];
                out_clusters_size[blockIdx.x + j] = out_clusters_size[j];
            }
        }
    }
}

__global__ void updateArrays(int n_clusters, float *out_clusters_x, float *out_clusters_y, int *out_clusters_size, float *d_clusters_center_x, float *d_clusters_center_y, int *d_clusters_size) {
    for(int i = 0; i < n_clusters; i++) {
        d_clusters_center_x[i] = out_clusters_x[i];
        d_clusters_center_y[i] = out_clusters_y[i];
        d_clusters_size[i] = out_clusters_size[i];
    }
}

/**
 * @brief Calculates the new clusters centers.
 *
 */
__global__ void divideSumClusterPoints(int n_clusters, float *nclusters_center_x, float *nclusters_center_y, int *nclusters_size)
{
    // Index variable.
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the new clusters centers.
    if (i < n_clusters)
    {
        nclusters_center_x[i] /= nclusters_size[i];
        nclusters_center_y[i] /= nclusters_size[i];
    }
}

/**
 * @brief Calculates the new cluster assignment for each point.
 *
 */
__global__ void calculate_clusters(int n_points, int n_clusters, float *npoints_x, float *npoints_y, int *npoints_cluster, float *nclusters_center_x, float *nclusters_center_y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_points)
    {
        // Get the x and y coordinates of the i-th point.
        float x1 = npoints_x[i];
        float y1 = npoints_y[i];

        // Variable that will store the cluster assignment of the i-th point.
        // To better check errors, the point is "assigned" to cluster -1 (invalid).
        int cluster = -1;

        // Indicates that the distance between a point and a cluster center
        //  was not yet calculated.
        float distance = -1;

        for (int j = 0; j < n_clusters; j++)
        {
            // Get the x and y coordinates of the j-th cluster center.
            float x2 = nclusters_center_x[j];
            float y2 = nclusters_center_y[j];

            // Calculate the squared distance between the i-th point and the
            //  j-th cluster center.
            float distance_j = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1);

            // Check if the i-th point is closer to the j-th cluster center
            //  than the previous closest cluster center.
            // If the distance is -1, then this is the first cluster center
            //  checked and the i-th point is automatically assigned to it.
            if (distance_j < distance || distance < 0)
            {
                cluster = j;
                distance = distance_j;
            }
        }

        // Update the cluster assignment of the i-th point.
        if (npoints_cluster[i] != cluster)
        {
            // *changed = 1;
            npoints_cluster[i] = cluster;
        }
    }
}

/**
 * @brief Prints the results of the k-means algorithm.
 *        Prints each cluster center, its size and the number of iterations.
 */
void print_results(int n_points, int n_clusters, int iterations)
{
    // Index variable.
    int i;

    // Number of points and clusters.
    printf("N = %d, K = %d\n", n_points, n_clusters);

    // Center and number of points in each cluster.
    for (i = 0; i < n_clusters; i++)
    {
        printf("Center: (%.3f, %.3f) : Size: %d\n", clusters_center_x[i], clusters_center_y[i], clusters_size[i]);
    }

    // Number of iterations.
    printf("Iterations: %d\n", iterations);
}

/**
 * @brief K-means algorithm.
 *       The algorithm will stop when the clusters stop changing.
 *
 * @return int The number of iterations.
 */
int k_means(int n_points, int n_clusters)
{
    // Number of iterations.
    int iterations = 0;
    // int changed = 0;

    // Device grid dimensions
    int block_size = 512;
    int grid_size =( n_points + block_size) / block_size;
    int grid_size2 = (n_clusters + block_size) / block_size;

    printf("block num: %d\n", grid_size);

    // Device variables
    float *d_points_x, *d_points_y, *d_clusters_center_x, *d_clusters_center_y, *out_clusters_x, *out_clusters_y;
    int *d_points_cluster, *d_clusters_size, *out_clusters_size;

    // Allocate device memory
    hipMalloc((void **)&d_points_x, sizeof(float) * n_points);
    hipMalloc((void **)&d_points_y, sizeof(float) * n_points);
    hipMalloc((void **)&d_points_cluster, sizeof(int) * n_points);
    hipMalloc((void **)&d_clusters_center_x, sizeof(float) * n_clusters);
    hipMalloc((void **)&d_clusters_center_y, sizeof(float) * n_clusters);
    hipMalloc((void **)&d_clusters_size, sizeof(int) * n_clusters);
    hipMalloc((void **)&out_clusters_x, sizeof(float) * n_clusters * grid_size);
    hipMalloc((void **)&out_clusters_y, sizeof(float) * n_clusters * grid_size);
    hipMalloc((void **)&out_clusters_size, sizeof(int) * n_clusters * grid_size);

    // Transfer data from host to device memory
    hipMemcpy(d_points_x, points_x, sizeof(float) * n_points, hipMemcpyHostToDevice);
    hipMemcpy(d_points_y, points_y, sizeof(float) * n_points, hipMemcpyHostToDevice);
    hipMemcpy(d_points_cluster, points_cluster, sizeof(int) * n_points, hipMemcpyHostToDevice);
    hipMemcpy(d_clusters_center_x, clusters_center_x, sizeof(float) * n_clusters, hipMemcpyHostToDevice);
    hipMemcpy(d_clusters_center_y, clusters_center_y, sizeof(float) * n_clusters, hipMemcpyHostToDevice);

    // Calculate the clusters for the first time.
    calculate_clusters<<<grid_size, block_size>>>(n_points, n_clusters, d_points_x, d_points_y, d_points_cluster, d_clusters_center_x, d_clusters_center_y);

    // Keep calculating the clusters for 20 iterations
    // for (iterations = 0; iterations < 20; iterations++)
    // {
    //     equalToZero<<<grid_size2, block_size>>>(n_clusters, d_clusters_center_x, d_clusters_center_y, d_clusters_size);
    //     // sumClusterPoints<<<grid_size, block_size, 3 * block_size * n_clusters>>>(n_points, n_clusters, d_points_x, d_points_y, d_points_cluster, out_clusters_x, out_clusters_y, out_clusters_size);
    //     // reduceArrays<<<1,1>>>(grid_size, n_clusters, out_clusters_x, out_clusters_y, out_clusters_size, d_clusters_center_x, d_clusters_center_y, d_clusters_size);
    //     divideSumClusterPoints<<<grid_size2, block_size>>>(n_clusters, d_clusters_center_x, d_clusters_center_y, d_clusters_size);

    //     // changed = 0;
    //     calculate_clusters<<<grid_size, block_size>>>(n_points, n_clusters, d_points_x, d_points_y, d_points_cluster, d_clusters_center_x, d_clusters_center_y);
    // }

    equalToZero<<<grid_size2, block_size>>>(n_clusters, d_clusters_center_x, d_clusters_center_y, d_clusters_size);
    sumClusterPoints<<<grid_size, block_size, 3 * block_size * n_clusters>>>(n_points, n_clusters, d_points_x, d_points_y, d_points_cluster, out_clusters_x, out_clusters_y, out_clusters_size);
    
    // int blockNum = grid_size;
    // // while(blockNum > block_size) {
    // //     int newGrid = (blockNum + block_size) / block_size;
    // //     reduceArrays<<<newGrid,block_size>>>(blockNum, n_clusters, out_clusters_x, out_clusters_y, out_clusters_size);
    // //     blockNum = newGrid;
    // // }
    // reduceArrays<<<1,block_size>>>(blockNum, n_clusters, out_clusters_x, out_clusters_y, out_clusters_size);
    // updateArrays<<<1,1>>>(n_clusters, out_clusters_x, out_clusters_y, out_clusters_size, d_clusters_center_x, d_clusters_center_y, d_clusters_size);
    // divideSumClusterPoints<<<grid_size2, block_size>>>(n_clusters, d_clusters_center_x, d_clusters_center_y, d_clusters_size);

    hipMemcpy(clusters_center_x, d_clusters_center_x, sizeof(float) * n_clusters, hipMemcpyDeviceToHost);
    hipMemcpy(clusters_center_y, d_clusters_center_y, sizeof(float) * n_clusters, hipMemcpyDeviceToHost);
    hipMemcpy(clusters_size, d_clusters_size, sizeof(int) * n_clusters, hipMemcpyDeviceToHost);


    // Deallocate device memory
    hipFree(d_points_x);
    hipFree(d_points_y);
    hipFree(d_points_cluster);
    hipFree(d_clusters_center_x);
    hipFree(d_clusters_center_y);

    return iterations;
}

/**
 * @brief Main function.
 *        It initializes the data and calls the k-means algorithm.
 *        It also prints the results and frees the allocated memory.
 *
 * Needed arguments:
 * - Number of points.
 * - Number of clusters.
 * - Number of threads.
 *
 * @return int 0 if the program finishes successfully.
 */
int main(int argc, char *argv[])
{

    // Declare the event handles
    hipEvent_t start, stop;

    // Create the events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start time
    hipEventRecord(start, 0);

    // Variables to store the number of points, clusters and threads.
    int n_points, n_clusters;

    // Check if the number of arguments is correct.
    if (argc < 3)
    {
        printf("Usage: %s <number of points> <number of clusters> <number of threads>", argv[0]);
        exit(1);
    }
    else
    {
        n_points = atoi(argv[1]);
        n_clusters = atoi(argv[2]);

        // Print command line arguments.
        // printf("Points: %d | Clusters: %d | Threads: %d \n", n_points, n_clusters, n_threads);
    }

    // Initialize the data structures.
    init_ds(n_points, n_clusters);

    // Call the k-means algorithm.
    int iterations = k_means(n_points, n_clusters);

    // Show the results.
    print_results(n_points, n_clusters, iterations);

    // Free the allocated memory.
    free_data();

    // Record the stop time
    hipEventRecord(stop, 0);

    // Wait for the events to complete
    hipEventSynchronize(start);
    hipEventSynchronize(stop);

    // Get the elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // Print the elapsed time in milliseconds
    printf("Elapsed time: %f ms\n", elapsedTime);

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
